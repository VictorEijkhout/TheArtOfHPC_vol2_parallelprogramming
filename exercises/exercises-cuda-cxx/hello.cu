// -*- c -*-
/****************************************************************
 ****
 **** This file belongs with the course
 **** Parallel Programming in MPI and OpenMP
 **** copyright 2016-2025 Victor Eijkhout eijkhout@tacc.utexas.edu
 ****
 **** hello.cxx : single thread CUDA hello world
 ****
 ****************************************************************/

#include "hip/hip_runtime.h"


#include <stdio.h>

//codesnippet cuhellodef
__global__ void hello_cuda() {
  printf("hello world!\n");
}
//codesnippet end

int main() {
  //codesnippet cuhellouse
  hello_cuda<<<1,1>>>();
  hipDeviceSynchronize();
  //codesnippet end

  return 0;
}
