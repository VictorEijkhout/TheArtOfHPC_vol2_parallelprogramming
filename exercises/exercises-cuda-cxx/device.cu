#include "hip/hip_runtime.h"
// -*- c++ -*-
/****************************************************************
 ****
 **** This file belongs with the course
 **** Parallel Programming in MPI and OpenMP
 **** copyright 2016-2025 Victor Eijkhout eijkhout@tacc.utexas.edu
 ****
 **** device.cxx : dealing with cuda devices
 ****
 ****************************************************************/

#include <iostream>
using std::cout;

int main() {

  const int ndev = 
    [] () ->int {
      //codesnippet cudevcount
      int ndev;
      auto status = hipGetDeviceCount(&ndev);
      //codesnippet end
      if (status==hipSuccess)
	return ndev;
      else if (status==hipErrorNoDevice) {
	cout << "No devices found\n"; throw status;
      } else if (status==hipErrorInsufficientDriver) {
	cout << "Insufficient driver\n"; throw status;
      } else return 0; }();

  cout << "Number of devices detected: " << ndev << '\n';
  for ( int idev=0; idev<ndev; ++idev ) {
    //codesnippet cudevprop
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties,idev);
    //codesnippet end
    //codesnippet cudevprops
    cout << "Device " << idev << "=" << properties.name << '\n';
    cout << "  async: " << properties.asyncEngineCount << '\n';
    cout << "  unified: " << properties.unifiedAddressing << '\n';

    cout << "  capability: " << properties.major << "." << properties.minor << '\n';
    cout << "  multiprocs: " << properties.multiProcessorCount << '\n';
    cout << "  clock rate: " << properties.clockRate << '\n';

    cout << "  global memory: " << properties.totalGlobalMem << '\n';
    cout << "  shared mem/block: " << properties.sharedMemPerBlock << '\n';

    cout << "  max threads/block: " << properties.maxThreadsPerBlock << '\n';
    cout << "  max thread dims: " << properties.maxThreadsDim[0] << ","
	 << properties.maxThreadsDim[1] << "," 
	 << properties.maxThreadsDim[2] << '\n';
    //codesnippet end
  }

#if CUDA_VERSION >= 12000
  cudaInitDevice(0,0,0);
#endif

  return 0;
}
