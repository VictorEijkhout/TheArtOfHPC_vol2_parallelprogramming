#include "hip/hip_runtime.h"
// -*- c -*-

#include <iostream>
#include <math.h>
#include <chrono>
using namespace std::chrono;
using myclock = steady_clock;
#include "cxxopts.hpp"

// function to add the elements of two arrays
__global__
void add1(size_t n, float *x, float *y, float *z)
{
  size_t t_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (t_id>=n) return;
  z[t_id] = x[t_id] + y[t_id];
}

#define CU_ASSERT( code ) { \
  hipError_t err = code ; \
  if (err!=hipSuccess) { \
    printf("error <<%s>> on line %d\n",hipGetErrorString(err),__LINE__); \
    return 1; } }

int main(int argc,char **argv) {

  cxxopts::Options options
    ( "cxxopts","Add 3 vectors with managed memory" );
  options.add_options()
    ( "h,help","usage information" )
    ( "n,nsize","size of vector", cxxopts::value<size_t>()->default_value("100") );
  auto result = options.parse(argc, argv);
  if (result.count("help")>0) {
    std::cout << options.help() << '\n';
    return 0;
  }
  /* } catch ( cxxopts::exceptions::option_has_no_value ) { */
  /*   std::cout << "Option has no value" << '\n'; */
  /* } */


  auto N = result["nsize"].as<size_t>();
  printf("Running with N=%ld\n",N);

  float
    *x, *y, *z;
  size_t nbytes = N*sizeof(float);
  CU_ASSERT( hipMallocManaged( &x,nbytes ) );
  CU_ASSERT( hipMallocManaged( &y,nbytes ) );
  CU_ASSERT( hipMallocManaged( &z,nbytes ) );

  // initialize x and y arrays on the host
  for (size_t i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
    z[i] = 0.0f;
  }

  myclock::time_point before =
    myclock::now();

  // Run kernel on 1M elements on the CPU
  int blocksize = 1024;
  int gridsize = N/blocksize+1;
  add1<<<gridsize,blocksize>>>(N,x,y,z);
  // Wait for GPU to finish before accessing data on host
  CU_ASSERT( hipDeviceSynchronize() );

  auto after = myclock::now();
  std::cout << "Action took: "
       << duration_cast<milliseconds>
            (after-before).count()
       << "ms\n";

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    //printf("%4.1f",z[i]);
    if (z[i]!=3.0f) {
      printf("error at %d\n",i);
      break;
    }
    maxError = fmax(maxError, fabs(z[i]-3.0f));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  // delete [] x;
  // delete [] y;
  hipFree(x);
  hipFree(y);

  return 0;
}
